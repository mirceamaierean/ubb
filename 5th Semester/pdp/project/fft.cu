#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <chrono>

// Define M_PI if not already defined
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

// Define CHECK_CUDA for error handling
#define CHECK_CUDA(stat)                                               \
    if ((stat) != hipSuccess)                                         \
    {                                                                  \
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(stat)); \
        exit(EXIT_FAILURE);                                            \
    }

// Define reverse_bits_gpu for bit-reversal
__device__ uint32_t reverse_bits_gpu(uint32_t x)
{
    x = ((x & 0xaaaaaaaa) >> 1) | ((x & 0x55555555) << 1);
    x = ((x & 0xcccccccc) >> 2) | ((x & 0x33333333) << 2);
    x = ((x & 0xf0f0f0f0) >> 4) | ((x & 0x0f0f0f0f) << 4);
    x = ((x & 0xff00ff00) >> 8) | ((x & 0x00ff00ff) << 8);
    return (x >> 16) | (x << 16);
}

using namespace std;
using namespace std::chrono;

const double PI = acos(-1);

// FFT Kernel with inverse flag
__global__ void fft_kernel(const hipFloatComplex *x, hipFloatComplex *Y, uint32_t N, int logN, bool inverse)
{
    uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i >= N)
        return;

    // Bit-reversal permutation
    uint32_t rev = reverse_bits_gpu(i);
    rev = rev >> (32 - logN);
    Y[i] = x[rev];

    __syncthreads();

    // Cooley-Tukey FFT
    for (int s = 1; s <= logN; s++)
    {
        int mh = 1 << (s - 1); // Half-size
        int m = 1 << s;        // Size
        int k = (i / mh) * m;  // Block start
        int j = i % mh;        // Position within block

        if (j < mh)
        {
            // Compute the twiddle factor
            float angle = (inverse ? +1.0f : -1.0f) * (float)M_PI * j / mh;
            float tr, ti;
            sincosf(angle, &ti, &tr);
            hipFloatComplex twiddle = make_hipFloatComplex(tr, ti);

            // Butterfly operations
            hipFloatComplex a = Y[k + j];
            hipFloatComplex b = hipCmulf(twiddle, Y[k + j + mh]);

            Y[k + j] = hipCaddf(a, b);
            Y[k + j + mh] = hipCsubf(a, b);
        }

        __syncthreads();
    }
}

// FFT GPU Function
int fft_gpu(const hipFloatComplex *x, hipFloatComplex *Y, uint32_t N, bool inverse)
{
    if (N & (N - 1))
    {
        fprintf(stderr, "N=%u must be a power of 2.\n", N);
        return -1;
    }

    int logN = (int)log2f((float)N);
    hipError_t st;

    hipFloatComplex *x_dev;
    hipFloatComplex *Y_dev;

    // Allocate device memory
    st = hipMalloc((void **)&Y_dev, sizeof(*Y) * N);
    CHECK_CUDA(st);

    st = hipMalloc((void **)&x_dev, sizeof(*x) * N);
    CHECK_CUDA(st);

    // Copy input data to device
    st = hipMemcpy(x_dev, x, sizeof(*x) * N, hipMemcpyHostToDevice);
    CHECK_CUDA(st);

    // Define CUDA grid and block dimensions
    int block_size = 256;
    dim3 block(block_size, 1);
    dim3 grid((N + block_size - 1) / block_size, 1);

    // Launch FFT kernel
    fft_kernel<<<grid, block>>>(x_dev, Y_dev, N, logN, inverse);
    st = hipGetLastError();
    CHECK_CUDA(st);

    // Copy result back to host
    st = hipMemcpy(Y, Y_dev, sizeof(*x) * N, hipMemcpyDeviceToHost);
    CHECK_CUDA(st);

    // Free device memory
    st = hipFree(x_dev);
    CHECK_CUDA(st);
    st = hipFree(Y_dev);
    CHECK_CUDA(st);

    // Normalize if inverse FFT
    if (inverse)
    {
        for (uint32_t i = 0; i < N; i++)
        {
            Y[i].x /= N;
            Y[i].y /= N;
        }
    }

    return 0;
}

// Bit-Reversal Function for Host
uint32_t reverse_bits(uint32_t x)
{
    x = ((x & 0xaaaaaaaa) >> 1) | ((x & 0x55555555) << 1);
    x = ((x & 0xcccccccc) >> 2) | ((x & 0x33333333) << 2);
    x = ((x & 0xf0f0f0f0) >> 4) | ((x & 0x0f0f0f0f) << 4);
    x = ((x & 0xff00ff00) >> 8) | ((x & 0x00ff00ff) << 8);
    return (x >> 16) | (x << 16);
}

// Function to Clean Coefficients by Removing Small Artifacts
vector<long long> clean_coefficients(const vector<hipFloatComplex> &data, int n, double threshold = 1e-3)
{
    vector<long long> result(n, 0);
    for (int i = 0; i < n; i++)
    {
        float real_part = hipCrealf(data[i]);
        if (fabs(real_part) > threshold)
        {
            result[i] = round(real_part);
        }
    }
    return result;
}

// Debugging Function to Print Complex Vectors
void debug_output(const vector<hipFloatComplex> &data, const string &label)
{
    cout << label << ":\n";
    for (size_t i = 0; i < data.size(); ++i)
    {
        cout << "Index: " << i << ", Value: (" << hipCrealf(data[i]) << ", " << hipCimagf(data[i]) << ")\n";
    }
    cout << endl;
}

// Debugging Function to Print Zero-Padded Arrays
void debug_zero_padding(const vector<hipFloatComplex> &data, const string &label)
{
    cout << label << " (size = " << data.size() << "):\n";
    for (size_t i = 0; i < data.size(); i++)
    {
        cout << "Index: " << i << ", Value: (" << hipCrealf(data[i]) << ", " << hipCimagf(data[i]) << ")\n";
    }
    cout << endl;
}

// Debugging Function to Print Final Cleaned Result
void debug_final_result(const vector<long long> &result)
{
    cout << "Final cleaned result: ";
    for (size_t i = 0; i < result.size(); i++)
    {
        cout << result[i] << " ";
    }
    cout << endl;
}

// Polynomial Multiplication Function
vector<long long> multiply(const vector<long long> &a, const vector<long long> &b)
{
    // Determine the size for FFT (next power of 2)
    int n = 1;
    while (n < a.size() + b.size())
    {
        n <<= 1;
    }

    // Initialize complex vectors with zero-padding
    vector<hipFloatComplex> fa(n, make_hipFloatComplex(0.0, 0.0));
    vector<hipFloatComplex> fb(n, make_hipFloatComplex(0.0, 0.0));

    // Populate fa with coefficients of polynomial a
    for (size_t i = 0; i < a.size(); ++i)
    {
        fa[i] = make_hipFloatComplex(a[i], 0.0);
    }

    // Populate fb with coefficients of polynomial b
    for (size_t i = 0; i < b.size(); ++i)
    {
        fb[i] = make_hipFloatComplex(b[i], 0.0);
    }

    // Perform forward FFT on both polynomials
    vector<hipFloatComplex> fa_transformed(n);
    vector<hipFloatComplex> fb_transformed(n);

    fft_gpu(fa.data(), fa_transformed.data(), n, false);
    fft_gpu(fb.data(), fb_transformed.data(), n, false);

    // Element-wise multiplication in frequency domain
    vector<hipFloatComplex> result_transformed(n);
    for (int i = 0; i < n; i++)
    {
        result_transformed[i] = hipCmulf(fa_transformed[i], fb_transformed[i]);
    }

    // Perform inverse FFT on the multiplied result
    vector<hipFloatComplex> fa_inverse(n);
    fft_gpu(result_transformed.data(), fa_inverse.data(), n, true);

    // Clean coefficients by removing small numerical artifacts
    vector<long long> result = clean_coefficients(fa_inverse, n);

    // Trim trailing zeros to get the final polynomial coefficients
    while (!result.empty() && result.back() == 0)
    {
        result.pop_back();
    }

    // Debug final cleaned result
    debug_final_result(result);

    return result;
}

// Main Function
int main()
{
    // Define two polynomials a(x) and b(x)
    // Example:
    // a(x) = 9 + x + 3x^2 + 7x^3 + 5x^4 + 4x^5
    // b(x) = 7 + 8x + 8x^2 + 3x^3 + 6x^4 + 2x^5
    ifstream fin("File1.txt");
    ofstream fout("ok.txt");
    vector<long long> a, b;
    fin >> n;
    for (int i = 0; i <= n; i++)
    {
        int x;
        fin >> x;
        a.push_back(x);
    }
    fin >> m;
    for (int i = 0; i <= m; i++)
    {
        int x;
        fin >> x;
        b.push_back(x);
    }

    auto start = high_resolution_clock::now();

    // Perform polynomial multiplication
    vector<long long> c = multiply(a, b);

    auto end = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(end - start);

    cout << "FFT Cuda Execution Time: " << duration.count() << " microseconds" << endl;

    for (int i = 0; i < c.size(); i++)
    {
        fout << c[i] << " ";
    }
    fout << endl;
    return 0;
}
